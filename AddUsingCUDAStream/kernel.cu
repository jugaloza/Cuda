#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#define N 50000

__global__ void addWithCUDA(int* d_a, int* d_b, int* d_c)
{
    int threadId = threadIdx.x + (blockDim.x * blockIdx.x);

    while (threadId < N)
    {
        d_c[threadId] = d_a[threadId] + d_b[threadId];
        threadId += blockDim.x * gridDim.x;
    }
}

int main()
{
    
    int* h_a, * h_b, * h_c;

    int* d_a0, *d_a1, *d_b0, *d_b1;
    int* d_c0, * d_c1;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CUDASetDevice failed. " << std::endl;
        return -1;
    }

    hipStream_t stream0, stream1;

    cudaStatus = hipStreamCreate(&stream0);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "hipStreamCreate failed. " << std::endl;
        return -1;
    }
    
    cudaStatus = hipStreamCreate(&stream1);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "hipStreamCreate failed. " << std::endl;
        return -1;
    }

    hipEvent_t e_start, e_stop;

    cudaStatus = hipEventCreate(&e_start);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "hipEventCreate failed. " << std::endl;
        return -1;
    }

    cudaStatus = hipEventCreate(&e_stop);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "hipEventCreate failed. " << std::endl;
        return -1;
    }

    cudaStatus = hipEventRecord(e_start, 0);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaEventRecord failed. " << std::endl;
        return -1;
    }

    cudaStatus = hipHostAlloc((void**)&h_a, 2 * N * sizeof(int), hipHostMallocDefault);
    
    if (cudaStatus != hipSuccess)
    {
        std::cout << "hipHostAlloc failed. " << std::endl;
        return -1;
    }

    cudaStatus = hipHostAlloc((void**)&h_b, 2 * N * sizeof(int), hipHostMallocDefault);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "hipHostAlloc failed. " << std::endl;
        return -1;
    }

    cudaStatus = hipHostAlloc((void**)&h_c, 2 * N * sizeof(int), hipHostMallocDefault);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "hipHostAlloc failed. " << std::endl;
        return -1;
    }

    cudaStatus = hipMalloc((void**)&d_a0, N * sizeof(int));

    if (cudaStatus != hipSuccess)
    {
        std::cout << "hipMalloc failed. " << std::endl;
        return -1;
    }

    cudaStatus = hipMalloc((void**)&d_a1, N * sizeof(int));

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMalloc failed. " << std::endl;
        return -1;
    }

    cudaStatus = hipMalloc((void**)&d_b0, N * sizeof(int));

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMalloc failed. " << std::endl;
        return -1;
    }

    cudaStatus = hipMalloc((void**)&d_b1, N * sizeof(int));

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMalloc failed. " << std::endl;
        return -1;
    }

    cudaStatus = hipMalloc((void**)&d_c0, N * sizeof(int));

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMalloc failed. " << std::endl;
        return -1;
    }

    cudaStatus = hipMalloc((void**)&d_c1, N * sizeof(int));

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMalloc failed. " << std::endl;
        return -1;
    }

    for (int idx = 0; idx < 2 * N; idx++)
    {
        h_a[idx] = 2 * idx * idx;
        h_b[idx] = idx;
    }

    cudaStatus = hipMemcpyAsync(d_a0, h_a, N * sizeof(int), hipMemcpyHostToDevice, stream0);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMemcpyAsync failed. " << std::endl;
        hipFree(d_a0);
        hipFree(d_a1);
        hipFree(d_b0);
        hipFree(d_b1);
        hipFree(d_c0);
        hipFree(d_c1);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        return -1;
    }
    
    cudaStatus = hipMemcpyAsync(d_a1, h_a + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMemcpyAsync failed. " << std::endl;
        hipFree(d_a0);
        hipFree(d_a1);
        hipFree(d_b0);
        hipFree(d_b1);
        hipFree(d_c0);
        hipFree(d_c1);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        return -1;
    }

    cudaStatus = hipMemcpyAsync(d_b0, h_b, N * sizeof(int), hipMemcpyHostToDevice, stream0);


    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMemcpyAsync failed. " << std::endl;
        hipFree(d_a0);
        hipFree(d_a1);
        hipFree(d_b0);
        hipFree(d_b1);
        hipFree(d_c0);
        hipFree(d_c1);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        return -1;
    }

    cudaStatus = hipMemcpyAsync(d_b1, h_b + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMemcpyAsync failed. " << std::endl;
        hipFree(d_a0);
        hipFree(d_a1);
        hipFree(d_b0);
        hipFree(d_b1);
        hipFree(d_c0);
        hipFree(d_c1);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        return -1;
    }

    addWithCUDA << < 512, 512, 0, stream0 >> > (d_a0, d_b0, d_c0);
    addWithCUDA << < 512, 512, 0, stream1 >> > (d_a1, d_b1, d_c1);

    cudaStatus = hipMemcpyAsync(h_c, d_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMemcpyAsync failed. " << std::endl;
        hipFree(d_a0);
        hipFree(d_a1);
        hipFree(d_b0);
        hipFree(d_b1);
        hipFree(d_c0);
        hipFree(d_c1);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        return -1;
    }

    cudaStatus = hipMemcpyAsync(h_c + N, d_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMemcpyAsync failed. " << std::endl;
        hipFree(d_a0);
        hipFree(d_a1);
        hipFree(d_b0);
        hipFree(d_b1);
        hipFree(d_c0);
        hipFree(d_c1);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        return -1;
    }

    cudaStatus = hipDeviceSynchronize();

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMemcpyAsync failed. " << std::endl;
        hipFree(d_a0);
        hipFree(d_a1);
        hipFree(d_b0);
        hipFree(d_b1);
        hipFree(d_c0);
        hipFree(d_c1);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        return -1;
    }

    cudaStatus = hipStreamSynchronize(stream0);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMemcpyAsync failed. " << std::endl;
        hipFree(d_a0);
        hipFree(d_a1);
        hipFree(d_b0);
        hipFree(d_b1);
        hipFree(d_c0);
        hipFree(d_c1);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        return -1;
    }

    cudaStatus = hipStreamSynchronize(stream1);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMemcpyAsync failed. " << std::endl;
        hipFree(d_a0);
        hipFree(d_a1);
        hipFree(d_b0);
        hipFree(d_b1);
        hipFree(d_c0);
        hipFree(d_c1);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        return -1;
    }

    cudaStatus  = hipEventRecord(e_stop, 0);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMemcpyAsync failed. " << std::endl;
        hipFree(d_a0);
        hipFree(d_a1);
        hipFree(d_b0);
        hipFree(d_b1);
        hipFree(d_c0);
        hipFree(d_c1);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        return -1;
    }

    cudaStatus = hipEventSynchronize(e_stop);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMemcpyAsync failed. " << std::endl;
        hipFree(d_a0);
        hipFree(d_a1);
        hipFree(d_b0);
        hipFree(d_b1);
        hipFree(d_c0);
        hipFree(d_c1);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        return -1;
    }

    float elapsed_time;

    hipEventElapsedTime(&elapsed_time, e_start, e_stop);

    std::cout << "Elapsed Time : " << elapsed_time << " ms " << std::endl;

    bool correctSum = 1;

    for (int idx = 0; idx < 2 * N; idx++)
    {
        if (h_a[idx] + h_b[idx] != h_c[idx])
        {
            correctSum = 0;
        }
    }
    
    if (!correctSum)
    {
        std::cout << "GPU sum is not computed correctly.  " << std::endl;
    }
    else
    {
       std::cout << "GPU sum is  computed correctly.  " << std::endl;
    }

    hipFree(d_a0);
    hipFree(d_a1);
    hipFree(d_b0);
    hipFree(d_a1);
    hipFree(d_c0);
    hipFree(d_c1);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    return 0;
}

