#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <iostream>

#define IMAGE_WIDTH 16
#define IMAGE_HEIGHT 16
#define MAX_PIXEL_INTENSITY 255
#define MIN_PIXEL_INTENSITY 0


__global__ void enhanceImageWithCuda(int* img, int* res_img,int scale, int val)
{
    unsigned int row_idx = (blockDim.y * blockIdx.y) + threadIdx.y;
    unsigned int col_idx = (blockDim.x * blockIdx.x) + threadIdx.x;

    res_img[col_idx * IMAGE_WIDTH + row_idx] = scale * img[col_idx * IMAGE_WIDTH + row_idx] + val;

}

int main()
{

    int img[IMAGE_HEIGHT][IMAGE_WIDTH] = { 0 };
    int resImg[IMAGE_HEIGHT][IMAGE_WIDTH];
    //fill array
    for (int h_idx = 0; h_idx < IMAGE_HEIGHT; h_idx++)
    {
        for (int w_idx = 0; w_idx < IMAGE_WIDTH; w_idx++)
        {
            img[h_idx][w_idx] = std::rand() % MAX_PIXEL_INTENSITY;
        }
    }

    int* d_resImg, * d_img;

    hipError_t errorStatus;

    errorStatus = hipSetDevice(0);

    if (errorStatus != hipSuccess)
    {
        std::cout << "Unable to find CUDA Capable device. please check if cuda capable device is available" << std::endl;
        return -1;
    }

    errorStatus = hipDeviceReset();

    if (errorStatus != hipSuccess)
    {
        std::cout << "Cuda Error : " << hipGetErrorString(errorStatus) << std::endl;
        return -1;
    }

    errorStatus = hipMalloc((void**)&d_img, sizeof(int) * (IMAGE_HEIGHT * IMAGE_WIDTH));

    if (errorStatus != hipSuccess)
    {
        std::cout << "Unable to allocate memory on device " << std::endl;
        return -1;
    }

    errorStatus = hipMalloc((void**)&d_resImg, sizeof(int) * (IMAGE_HEIGHT * IMAGE_WIDTH));

    if (errorStatus != hipSuccess)
    {
        std::cout << "Unable to allocate memory on device " << std::endl;
        return -1;
    }


    errorStatus = hipMemcpy(d_img, img, sizeof(int) * (IMAGE_WIDTH * IMAGE_HEIGHT), hipMemcpyHostToDevice);

    if (errorStatus != hipSuccess)
    {
        std::cout << "unable to copy content of input image from host to device " << std::endl;
        hipFree(d_img);
        hipFree(d_resImg);
        return -1;
    }

    dim3 numThreads(8, 8);
    dim3 numBlocks(IMAGE_WIDTH / numThreads.x, IMAGE_HEIGHT / numThreads.y);

    enhanceImageWithCuda << <numBlocks, numThreads >> > (d_img, d_resImg, 1, 2);

    errorStatus = hipDeviceSynchronize();

    if (errorStatus != hipSuccess)
    {
        std::cout << "Error  : " << hipGetErrorString(errorStatus) << std::endl;
        hipFree(d_img);
        hipFree(d_resImg);
        return -1;
    }


    errorStatus = hipMemcpy(resImg, d_resImg, sizeof(int) * (IMAGE_WIDTH * IMAGE_HEIGHT), hipMemcpyDeviceToHost);

    if (errorStatus != hipSuccess)
    {
        std::cout << "Error unable to copy content of device to host " << std::endl;;
        hipFree(d_img);
        hipFree(d_resImg);
        return -1;
    }

    hipFree(d_img);
    hipFree(d_resImg);

    for (int h_idx = 0; h_idx < IMAGE_HEIGHT; h_idx++)
    {
        for (int w_idx = 0; w_idx < IMAGE_WIDTH; w_idx++)
        {
            std::cout << resImg[h_idx][w_idx] << std::endl;

        }
    }
    
    std::cin.get();

    return 0;
}
