#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

__global__ void cubeKernelCuda(int* d_inp, int* d_out)
{
    unsigned int threadId = threadIdx.x +(blockDim.x*blockIdx.x);

    int val = d_inp[threadId];

    d_out[threadId] = val * val * val;

}


int main()
{
    const int N = 50;

    int h_inp[N] = { 0 };
    int h_out[N] = { 0 };

    //filling array
    for (int idx = 0; idx < N; idx++)
    {
        h_inp[idx] = idx;
    }

    int* d_inp;
    int* d_out;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess)
    {
        std::cout << " Unable to find CUDA capable device. " << hipGetErrorString(cudaStatus) << std::endl;
        return -1;
    }

    cudaStatus = hipMalloc((void**)&d_inp, sizeof(int) * N);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMalloc Failed for input array. " << hipGetErrorString(cudaStatus) << std::endl;
        return -1;
    }

    cudaStatus = hipMalloc((void**)&d_out, sizeof(int) * N);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMalloc Failed for output array. " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_inp);
        return -1;
    }

    cudaStatus = hipMemcpy(d_inp, h_inp, sizeof(int) * N, hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMemCpy Failed for input array from host to device. " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_inp);
        hipFree(d_out);
        return -1;
    }

    cubeKernelCuda << <2, 25 >> > (d_inp, d_out);

    cudaStatus = hipGetLastError();

    if (cudaStatus != hipSuccess)
    {
        std::cout << "Unable to complete kernel call on device. " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_inp);
        hipFree(d_out);
        return -1;
    }

    cudaStatus = hipDeviceSynchronize();

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaDeviceSynchronize failed : " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_inp);
        hipFree(d_out);
        return -1;
    }

    cudaStatus = hipMemcpy(h_out, d_out, sizeof(int) * N, hipMemcpyDeviceToHost);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "CudaMemcpy failed for output array. ErrorString : " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_inp);
        hipFree(d_out);
        return -1;

    }

    //printing output array

    for (int idx = 0; idx < N; idx++)
    {
        std::cout << h_out[idx] << std::endl;
    }
    hipFree(d_inp);
    hipFree(d_out);

    cudaStatus = hipDeviceReset();

    if (cudaStatus != hipSuccess)
    {
        std::cout << "hipDeviceReset failed. Error : " << hipGetErrorString(cudaStatus) << std::endl;
        return -1;
    }

    return 0;
}

