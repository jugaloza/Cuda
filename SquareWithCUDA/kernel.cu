﻿
#include "hip/hip_runtime.h"

#include <iostream>

__global__ void squareKernel(int* d_inp, int* d_out)
{
    unsigned int threadId = threadIdx.x;

    auto temp = d_inp[threadId];
    d_out[threadId] = temp * temp;

}
int main()
{
    const int N = 5;

    int h_inp[N] = { 0 };
    int h_out[N] = { 0 };

    for (int idx = 0; idx < N; idx++)
    {
        h_inp[idx] = idx + 1;
    }

    int* d_inp;
    int* d_out;

    hipError_t cudaStatus;


    cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "Unable to find CUDA capable device " << std::endl;
        return -1;
    }

    cudaStatus = hipMalloc((void**)&d_inp, sizeof(int) * N);
    
    if (cudaStatus != hipSuccess)
    {
        std::cout << "Error while allocating memory on device" << hipGetErrorString(cudaStatus) << std::endl;
        return -1;
    }

    cudaStatus = hipMalloc((void**)&d_out, sizeof(int) * N);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "Error while allocating memory on device for output array : " << hipGetErrorString(cudaStatus) << std::endl;
        return -1;
    }

    cudaStatus = hipMemcpy(d_inp, h_inp, sizeof(int) * N, hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "Unable to copy content of array from host to device for input array  : " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_inp);
        hipFree(d_out);
        return -1;
    }

    squareKernel <<<1, 5>>> (d_inp, d_out);

    cudaStatus = hipGetLastError();

    if (cudaStatus != hipSuccess)
    {
        std::cout << "Error thrown by last thread : " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_inp);
        hipFree(d_out);
        return -1;
    }

    cudaStatus = hipDeviceSynchronize();

    if (cudaStatus != hipSuccess)
    {
        std::cout << "Unable to sync device and some task has failed while execution : " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_inp);
        hipFree(d_out);
        return -1;
    }

    cudaStatus = hipMemcpy(h_out, d_out, sizeof(int) * N, hipMemcpyDeviceToHost);

    if (cudaStatus != hipSuccess)
    {
        std::cout << "Unable to copy content of output array from device to host. " << std::endl;
        hipFree(d_inp);
        hipFree(d_out);
        return -1;
    }

    //printing squares of vector
    for (int idx = 0; idx < N; idx++)
    {
        std::cout << h_out[idx] << std::endl;
        
    }

    hipFree(d_inp);
    hipFree(d_out);

    cudaStatus = hipDeviceReset();

    if (cudaStatus != hipSuccess)
    {
        std::cout << "Unable to reset states of device " << std::endl;
        return -1;
    }

    std::cin.get();
    return 0;
}
