#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <stdio.h>
#include <iostream>
#include <format>
#include <string>

#define M 8
#define N 8


__global__ void transposeMatrixCuda(int* arr, int* outputArr)
{
    int indexY = blockDim.y * blockIdx.y + threadIdx.y;
    int indexX = blockDim.x * blockDim.x + threadIdx.x;

    int Index = indexY * N + indexX;
    int transposeIndex = indexX * M + indexY;

    outputArr[transposeIndex] = arr[Index];

   
}

int main()
{
    const int totalSize = M * N;
    
    //int arr[M][N] = { 0 };
    //int res[N][M];


    int* arr;
    int* res;

    arr = (int*)malloc(totalSize * sizeof(int));
    

    //filling array
    for (int i = 0; i < totalSize; i++)
    {
        arr[i] = std::rand();
    }

    res = (int*)malloc(totalSize * sizeof(int));

    int* d_inpArray;
    int* d_outputArray;

    hipError_t errorStatus;

    errorStatus = hipSetDevice(0);

    if (errorStatus != hipSuccess)
    {
        std::cout << "Unable to find CUDA capable device " << std::endl;
        return -1;
    }


    errorStatus = hipMalloc((void**)&d_inpArray, sizeof(int) * totalSize);

    if (errorStatus != hipSuccess)
    {
        std::cout << " Unable to allocate memory on GPU device for input Array" << std::endl;
        return -1;
    }

    errorStatus = hipMalloc((void**)&d_outputArray, sizeof(int) * totalSize);

    if (errorStatus != hipSuccess)
    {
        std::cout << " Unable to allocate memory on GPU device for output Array" << std::endl;
        return -1;
    }

    errorStatus = hipMemcpy(d_inpArray, arr, sizeof(int) * totalSize, hipMemcpyHostToDevice);

    if (errorStatus != hipSuccess)
    {
        std::cout << "Unable to copy content of input array from host to device " << std::endl;
        return -1;
    }
    
    //Transpose matrix
    dim3 numThreads(4, 4);
    dim3 numBlocks(2, 2);

    transposeMatrixCuda << < numBlocks, numThreads >> > (d_inpArray, d_outputArray);

    errorStatus = hipDeviceSynchronize();

    if (errorStatus != hipSuccess)
    {
        std::cout << " Error while executing kernel : " << hipGetErrorString(errorStatus) << std::endl;
        hipFree(d_inpArray);
        hipFree(d_outputArray);
        return -1;
    }

    errorStatus = hipMemcpy(res, d_outputArray, sizeof(int) * totalSize, hipMemcpyDeviceToHost);

    if (errorStatus != hipSuccess)
    {
        std::cout << " Unable to copy content of resultant array from device to host " << std::endl;
        hipFree(d_inpArray);
        hipFree(d_outputArray);
        return -1;
    }
    
    //printing result

    std::cout << "After transposing matrix " << std::endl;

    for (int idx = 0; idx < totalSize; idx++)
    {
        std::cout << res[idx] << std::endl;
    }

    errorStatus = hipDeviceReset();

    if (errorStatus != hipSuccess)
    {
        std::cout << "Failed to reset GPU device  " << std::endl;
        hipFree(d_inpArray);
        hipFree(d_outputArray);
        return -1;
    }

    hipFree(d_inpArray);
    hipFree(d_outputArray);

    return 0;
}

